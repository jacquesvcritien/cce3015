#include "hip/hip_runtime.h"

using namespace std;
#include <cassert>
#include <array>
#include <fstream>
#include <sstream>
#include "stdio.h"
#include "jbutil.h"

void saveOutput(float *ii, int rows, int cols, string filename){

	ofstream outputFile;
	filename = filename.substr(filename.find("/") + 1); 
	printf("%s\n", filename.c_str());
	string filename_to_save = "outputs/vectoroutput_"+filename;
	outputFile.open(filename_to_save);

	for(size_t row = 0; row < rows; row++){
		for(size_t col = 0; col < cols; col++){
			outputFile << ii[row * cols +col] << " ";
		}
		outputFile << endl;
	}

	cout << "Result written to file" << endl;

	outputFile.close();
}

//function to check if a string is a number
bool isNumber(string number)
{
    int i = 0;
    //flag for finding a '.'
    bool point = false;

    //for each character in string
    for (; number[i] != 0; i++)
    {
	//if '.'
	if(number[i] == '.'){
		//if '.' and already found '.' or is first character
		if(point || i==0){
			return false;
		}

		//set flag
		point = true;
	}
	else{
		if(!isdigit(number[i]))
			return false;
	}

    }
    return true;
}

__global__ void calculateColumnSums(int rows, int cols, float *ii, const float *a)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if(i < rows){
		for(int j=0; j < cols; j++){
			int index = i * cols + j;
			float prev_val = (j==0) ? 0 : ii[index-1];
			ii[index] = prev_val + a[index];
		}
	}
}

__global__ void calculateRowSums(int rows, int cols, float *ii, const float *a)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if(i < cols){
                for(int j=0; j < rows; j++){
                        int index = j * cols + i;
                        int prev_index = index - cols;
                        float prev_val = (j==0) ? 0 : ii[prev_index];
                        ii[index] = prev_val + ii[index];
                }
        }
}

int main(int argc, char *argv[])
{
	//check that file was passed
	if(argc < 2 ){
		printf("Please pass in a filename\n");
		return 1;
	}

	//get filename
	string filename = argv[1];
	bool save = true;
	
	if(argc == 3){
		save = argv[2] == "true" || argv[2] == "t";
	}
	
	//open file
	ifstream file (filename);

	//if not open
	if (!file.is_open())
	{
		printf("File not found\n");
		return 1;
	}

	string line;
	//read first line
	getline (file,line);

	//read word by word in line
	istringstream iss(line);
	string arg;

	//get rows
	iss >> arg;
	if(!isNumber(arg)){
		printf("Rows must be a correct number\n");
		return 1;
	}
	int const rows = stoi(arg);

	//get cols
	iss >> arg;
	if(!isNumber(arg)){
		printf("Columns must be a correct number\n");
		return 1;
	}
	int const cols = stoi(arg);

	const int size = rows * cols * sizeof(float);
	float *a, *ii;
	a=(float*)malloc(size);	
	ii=(float*)malloc(size);	

	//read every line
	int row_counter =0;
	while ( getline (file,line) )
	{
		istringstream iss(line);
		int col_counter =0;
		//read every value in each line
		while(iss >> arg)
		{
			// check if passed value is number
			if(!isNumber(arg)){
				printf("Cell values must be valid numbers\n");
				return 1;
			}

			a[row_counter * cols +  col_counter] = stof(arg);
			ii[row_counter * cols + col_counter] = 0;
			col_counter++;
		}

		//if not enough cols
		if(col_counter != cols){
			printf("Not all cell values were specified - columns\n");
			return 1;
		}
		row_counter++;
	}

	//if not enough rows
	if(row_counter != rows){
		printf("Not all cell values were specified - rows\n");
		return 1;
	}

	float *da, *dii;

	hipMalloc((void**)&da, size);
	hipMalloc((void**)&dii, size);

	// start timer
	double t = jbutil::gettime();	

	// Copy over input from host to device
	hipMemcpy(da, a, size, hipMemcpyHostToDevice);
	hipMemcpy(dii, ii, size, hipMemcpyHostToDevice);

	free(a);

	int threadsInBlocks = 128;
	const int nblocks = (rows + (threadsInBlocks-1)) / threadsInBlocks;
	calculateColumnSums<<<nblocks, 64>>>(rows, cols, dii, da);
	calculateRowSums<<<nblocks, 64>>>(rows, cols, dii, da);

	// Copy over output from device to host
	hipMemcpy(ii, dii, size, hipMemcpyDeviceToHost);

	// stop timer
	t = jbutil::gettime() - t;

	printf("Time taken: %fs\n", t);

	if(save){
		saveOutput(ii, rows, cols, filename);
	}

	free(ii);
	// Free device memory
	hipFree(da);
	hipFree(dii);

}
