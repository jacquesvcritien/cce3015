#include "hip/hip_runtime.h"

using namespace std;
#include <cassert>
#include <fstream>
#include <sstream>
#include "stdio.h"
#include "jbutil.h"

//function to check if a string is a number
bool isNumber(string number)
{
    int i = 0;
    //flag for finding a '.'
    bool point = false;

    //for each character in string
    for (; number[i] != 0; i++)
    {
	//if '.'
	if(number[i] == '.'){
		//if '.' and already found '.' or is first character
		if(point || i==0){
			return false;
		}

		//set flag
		point = true;
	}
	else{
		if(!isdigit(number[i]))
			return false;
	}

    }
    return true;
}



__global__ void calculateIntegralImage(int rows, int cols, float *ii, const float *a, int pitch)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if(i < rows){
		for(int j=0; j < cols; j++){
			int index = i * pitch + j;
			float prev_val = (j==0) ? 0 : ii[index-1];
			ii[index] = prev_val + a[index];
		}
	}

	__syncthreads();


	if(i < cols){
                for(int j=0; j < rows; j++){
                        int index = j * pitch + i;
                        int prev_index = (j-1) * pitch + i;
                        float prev_val = (j==0) ? 0 : ii[prev_index];
                        ii[index] = prev_val + ii[index];
                }
        }
}

int main(int argc, char *argv[])
{
	//check that file was passed
	if(argc < 2 ){
		printf("Please pass in a filename\n");
		return 1;
	}

	//get filename
	string filename = argv[1];
	//open file
	ifstream file (filename);

	//if not open
	if (!file.is_open())
	{
		printf("File not found\n");
		return 1;
	}

	string line;
	//read first line
	getline (file,line);

	//read word by word in line
	istringstream iss(line);
	string arg;

	//get rows
	iss >> arg;
	if(!isNumber(arg)){
		printf("Rows must be a correct number\n");
		return 1;
	}
	int const rows = stoi(arg);

	//get cols
	iss >> arg;
	if(!isNumber(arg)){
		printf("Columns must be a correct number\n");
		return 1;
	}
	int const cols = stoi(arg);


	float a[rows][cols], ii[rows][cols];

	//read every line
	int row_counter =0;
	while ( getline (file,line) )
	{
		istringstream iss(line);
		int col_counter =0;
		//read every value in each line
		while(iss >> arg)
		{
			// check if passed value is number
			if(!isNumber(arg)){
				printf("Cell values must be valid numbers\n");
				return 1;
			}

			a[row_counter][col_counter] = stof(arg);
			ii[row_counter][col_counter] = 0;
			col_counter++;
		}

		//if not enough cols
		if(col_counter != cols){
			printf("Not all cell values were specified - columns\n");
			return 1;
		}
		row_counter++;
	}

	//if not enough rows
	if(row_counter != rows){
		printf("Not all cell values were specified - rows\n");
		return 1;
	}


	printf("INPUT\n");
	 for(int i=0; i < rows;i++){
                for(int j=0; j < cols;j++){
                        printf("%f ", a[i][j]);
                }
                printf("\n");
        }

	const int rowsize = cols * sizeof(float);
	float *da, *dii;

	size_t pitch;
	hipMallocPitch((void**)&da, &pitch, rowsize, rows);
	hipMallocPitch((void**)&dii, &pitch, rowsize, rows);

	// start timer
	double t = jbutil::gettime();	

	// Copy over input from host to device
	hipMemcpy2D(da, pitch, a, rowsize, rowsize, rows, hipMemcpyHostToDevice);
	hipMemcpy2D(dii, pitch, ii, rowsize, rowsize, rows, hipMemcpyHostToDevice);

	int threadsInBlocks = 64;
	const int nblocks = (rows + (threadsInBlocks-1)) / threadsInBlocks;
	assert(pitch % sizeof(float) == 0);
	const int ipitch = pitch / sizeof(float);
	calculateIntegralImage<<<nblocks, 64>>>(rows, cols, dii, da, ipitch);

	// Copy over output from device to host
	hipMemcpy2D(ii, rowsize, dii, pitch, rowsize, rows, hipMemcpyDeviceToHost);

	// stop timer
	t = jbutil::gettime() - t;

	printf("\nOUTPUT\n");
	 for(int i=0; i < rows;i++){
                for(int j=0; j < cols;j++){
                        printf("%f ", ii[i][j]);
                }
                printf("\n");
        }

	printf("Time taken: %fs\n", t);

	// Free device memory
	hipFree(da);
	hipFree(dii);

}
